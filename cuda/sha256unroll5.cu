#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdbool.h>
#include <stdio.h>
#include "sha256.h"

#define dec(a) ((a)+48)
//If a<10 add '0' (48) else add 87 ('a'-10)
#define hex(a) ((a)+((a)<10?48:87))

__constant__  char dni[]="TRWAGMYFPDXBNJZSQVHLCKE";
__constant__  unsigned int C[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf,
		0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5, 0xd807aa98,
		0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7,
		0xc19bf174, 0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f,
		0x4a7484aa, 0x5cb0a9dc, 0x76f988da, 0x983e5152, 0xa831c66d, 0xb00327c8,
		0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967, 0x27b70a85,
		0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e,
		0x92722c85, 0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819,
		0xd6990624, 0xf40e3585, 0x106aa070, 0x19a4c116, 0x1e376c08, 0x2748774c,
		0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3, 0x748f82ee,
		0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7,
		0xc67178f2 };


__global__ void sha256cat(unsigned char* datain, unsigned char* strdataout) {
        register unsigned int context_state[8];
	register unsigned int shadowRegister[8];
	register unsigned int messageSchedule[16]; 
        for (long int letra=0; letra <23; letra++){
        long int hilo=(blockDim.y*blockIdx.y+threadIdx.y)*blockDim.x*gridDim.x +(blockDim.x*blockIdx.x+threadIdx.x);
        hilo=hilo;
        //long int hilo= ( blockIdx.x*blockDim.x + threadIdx.x);
        //int letra = hilo/(100*31*12);
        int fecha= hilo;//- letra*31*12*100;
        hilo+= (31*12*90)*letra;
        int year= fecha / (31*12);
        int resto=fecha - year*31*12;
        int mes = resto /31;
        int dia = resto -( mes * 31);
        mes++;
        dia++;
        year+=10;
        for (int j=0; j < 16; j++) { messageSchedule[j]=0x00000000;}
        messageSchedule[0] = (datain[0] << 24)|(datain[1] << 16)|(datain[2] << 8)|(datain[3]);
        messageSchedule[1] = (datain[4] << 24)|(dni[letra] << 16)|(datain[6] << 8)|(datain[7]);
        messageSchedule[2] = (dec(year/10) << 24)|(dec(year%10) << 16)|(dec(mes/10) << 8)|(dec(mes%10));
        messageSchedule[3] = (dec(dia/10) << 24)|(dec(dia%10) << 16)|(datain[14] << 8)|(datain[15]);
        messageSchedule[4] = (datain[16] << 24)|(datain[17] << 16)|(datain[18] << 8)|(0x00000080);

        messageSchedule[15]|= 0x00000000 | (19*8);
 
        context_state[0] = 0x6a09e667;
        context_state[1] = 0xbb67ae85;
        context_state[2] = 0x3c6ef372;
        context_state[3] = 0xa54ff53a;
        context_state[4] = 0x510e527f;
        context_state[5] = 0x9b05688c;
        context_state[6] = 0x1f83d9ab;
        context_state[7] = 0x5be0cd19;

                shadowRegister[0] = 0x6a09e667;
                shadowRegister[1] = 0xbb67ae85;
                shadowRegister[2] = 0x3c6ef372;
                shadowRegister[3] = 0xa54ff53a;
                shadowRegister[4] = 0x510e527f;
                shadowRegister[5] = 0x9b05688c;
                shadowRegister[6] = 0x1f83d9ab;
                shadowRegister[7] = 0x5be0cd19;

#pragma unroll 64
 for (int i = 0; i < 64; i++) {
        if (i>=16){     messageSchedule[i % 16] = sigma1(messageSchedule[(i - 2)%16])
                                + messageSchedule[(i - 7)%16] + sigma0(messageSchedule[(i - 15)%16])
                                + messageSchedule[(i - 16)%16];
         }
                unsigned int textRegister1 = shadowRegister[7]
                                + epsilon1(shadowRegister[4])
                                + choice(shadowRegister[4], shadowRegister[5],
                                                shadowRegister[6]) + C[i] + messageSchedule[i%16];
                unsigned int textRegister2 = epsilon0(shadowRegister[0])
                                + majority(shadowRegister[0], shadowRegister[1],
                                                shadowRegister[2]);
#pragma unroll 7
                for (int j = 7; j > 0; j--) {
                        shadowRegister[j] = shadowRegister[j - 1];
                }
                shadowRegister[0] = textRegister1 + textRegister2;
                shadowRegister[4] += textRegister1;
        }

#pragma unroll 1715
        for (int j = 0; j < 1715; j++) {   // here is the main loop

#pragma unroll 16
        for (int i = 0, j = 0; i < 16; i++, j += 4) {
                unsigned char jm = j >> 3;
                unsigned int acc =  context_state[jm] + shadowRegister[jm];
                unsigned char im = (j >> 1) & 0x0000003;
                unsigned char hm = (acc >> (24- im*8)) & 0x000000ff;
                messageSchedule[i] = (hex(hm >> 4) << 24) | (hex(hm %16) << 16);
                im= ((j+2) >> 1) & 0x0000003; 
                hm = (acc >> (24- im*8)) & 0x000000ff;
                messageSchedule[i]|= (hex(hm >> 4) << 8) | (hex(hm %16) );
        }

                context_state[0] = 0x6a09e667;
                context_state[1] = 0xbb67ae85;
                context_state[2] = 0x3c6ef372;
                context_state[3] = 0xa54ff53a;
                context_state[4] = 0x510e527f;
                context_state[5] = 0x9b05688c;
                context_state[6] = 0x1f83d9ab;
                context_state[7] = 0x5be0cd19;

                shadowRegister[0] = 0x6a09e667;
                shadowRegister[1] = 0xbb67ae85;
                shadowRegister[2] = 0x3c6ef372;
                shadowRegister[3] = 0xa54ff53a;
                shadowRegister[4] = 0x510e527f;
                shadowRegister[5] = 0x9b05688c;
                shadowRegister[6] = 0x1f83d9ab;
                shadowRegister[7] = 0x5be0cd19;


#pragma unroll 64
        for (int i = 0; i < 64; i++) {
                if (i >=16) {
                           messageSchedule[i % 16] = sigma1(messageSchedule[(i - 2)%16])
                                + messageSchedule[(i - 7)%16] + sigma0(messageSchedule[(i - 15)%16])
                                + messageSchedule[(i - 16)%16];
                }
                unsigned int textRegister1 = shadowRegister[7]
                                + epsilon1(shadowRegister[4])
                                + choice(shadowRegister[4], shadowRegister[5],
                                                shadowRegister[6]) + C[i] + messageSchedule[i%16];
                unsigned int textRegister2 = epsilon0(shadowRegister[0])
                                + majority(shadowRegister[0], shadowRegister[1],
                                                shadowRegister[2]);
#pragma unroll 7
                for (int j = 7; j > 0; j--) {
                        shadowRegister[j] = shadowRegister[j - 1];
                }
                shadowRegister[0] = textRegister1 + textRegister2;
                shadowRegister[4] += textRegister1;
        }

#pragma unroll  8
        for (int i = 0; i < 8; i++) {
                shadowRegister[i]+= context_state[i] ; //cd is the init constant here
                context_state[i]=shadowRegister[i];
        }

#pragma unroll 16
        for (int i=0; i < 16; i++) {
        //for (int i = 0, j = 0; i < 16; i++, j += 4) {
         //solo 62= 512>>8 y 0 = 0x80 son relevantes. Los demas 0
        messageSchedule[i]=0;
        }
        messageSchedule[0]= 0x80 << 24;
        messageSchedule[15]= (512 >> 8) <<8;
                
#pragma unroll 64
        for (int i = 0; i < 64; i++) {
             if (i>=16) {
                           messageSchedule[i % 16] = sigma1(messageSchedule[(i - 2)%16])
                                + messageSchedule[(i - 7)%16] + sigma0(messageSchedule[(i - 15)%16])
                                + messageSchedule[(i - 16)%16];
             }
                unsigned int textRegister1 = shadowRegister[7]
                                + epsilon1(shadowRegister[4])
                                + choice(shadowRegister[4], shadowRegister[5],
                                                shadowRegister[6]) + C[i] + messageSchedule[i%16];
                unsigned int textRegister2 = epsilon0(shadowRegister[0])
                                + majority(shadowRegister[0], shadowRegister[1],
                                                shadowRegister[2]);
#pragma unroll 7
                for (int j = 7; j > 0; j--) {
                        shadowRegister[j] = shadowRegister[j - 1];
                }
                shadowRegister[0] = textRegister1 + textRegister2;
                shadowRegister[4] += textRegister1;
        }

      } // EXIT MAIN LOOP


#pragma unroll 8
                for (int j = 0; j < 8; j++) {
                        unsigned int  acc = context_state[j] + shadowRegister[j];
#pragma unroll 4
                 for (int i = 0; i < 4; i++) {
                        unsigned char h= (acc >> (24 - i * 8)) & 0x000000ff;
                        strdataout[64*hilo+2*(i + 4 * j)]= hex(h >> 4 );
                        strdataout[64*hilo+2*(i + 4 * j)+1]= hex(h%16);                      
                }
                }

}
}

__device__ void sha256Transform(unsigned int * context_state, unsigned char * data) {
}


