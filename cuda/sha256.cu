#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdbool.h>
#include <stdio.h>
#include "sha256.h"

inline void __cudaCheck(hipError_t err, const char* file, int line) {
#ifndef NDEBUG
        if (err != hipSuccess) {
                fprintf(stderr, "%s(%d): CUDA error: %s\n", __FILE__, __LINE__,
                                hipGetErrorString(err));
                exit (EXIT_FAILURE);
        }
#endif
}


__constant__ unsigned int C[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf,
		0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5, 0xd807aa98,
		0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7,
		0xc19bf174, 0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f,
		0x4a7484aa, 0x5cb0a9dc, 0x76f988da, 0x983e5152, 0xa831c66d, 0xb00327c8,
		0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967, 0x27b70a85,
		0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e,
		0x92722c85, 0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819,
		0xd6990624, 0xf40e3585, 0x106aa070, 0x19a4c116, 0x1e376c08, 0x2748774c,
		0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3, 0x748f82ee,
		0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7,
		0xc67178f2 };

__constant__ char hex[]="0123456789abcdef";
__constant__ char dni[]="TRWAGMYFPDXBNJZSQVHLCKE";
__device__ void doubleHash(unsigned char* hash, unsigned char * expanded, int lenght) {
#pragma unroll 32
   for (int i=0; i<lenght;i++) {
          expanded[2*i]=hex[ hash[i]/16];
          expanded[2*i+1]=hex[hash[i]%16];
  }
 expanded[2*lenght]='\0';
}

__device__ void printHash(unsigned char* hash, int length) {
	for (int i = 0; i < length; i++) {
		printf("%02x", hash[i]);
	}
	printf("\n");
}


__global__ void sha256cat(unsigned char* datain, unsigned char* strdataout) {
        sha256Context context;
        unsigned char hash[32] ;
        unsigned char strdata[65];
        for (int letra=0; letra <23; letra ++) {
        long int hilo=(blockDim.y*blockIdx.y+threadIdx.y)*blockDim.x*gridDim.x +(blockDim.x*blockIdx.x+threadIdx.x);
        //long int hilo= ( blockIdx.x*blockDim.x + threadIdx.x);
        //int letra = hilo/(100*31*12);
        hilo = hilo /10 ; //para ver si ahora tarda 90 segundos o menos
        int fecha= hilo; //- letra*31*12*100;
        hilo=fecha+letra*(90*31*12);  //solo 90 el year!!
        int year= fecha / (31*12);
        int resto=fecha - year*31*12;
        int mes = resto /31;
        int dia = resto -( mes * 31);
        mes++;
        dia++;
        year+=10;
        sha256Init(&context);
        for (int j=0; j < 20; j++ ) { context.data[j]=datain[j];}
        context.data[5]=dni[letra];
        context.data[8]=hex[year/10];
        context.data[9]=hex[year%10]; 
        context.data[10]=hex[mes/10];
        context.data[11]=hex[mes%10];
        context.data[12]=hex[dia/10];
        context.data[13]=hex[dia%10];
        context.data[19]='\0';
        context.dataLength=19;
        context.bitLength[0]=19*8; //anticipando final
       // printf(".19s\n",&data[0]);        
        //unsigned char tst[]="00000P1996020108034";
        //if (year==96 && mes==2 && dia ==1 &&letra==8 ) {
        //     printf("%s %d\n",data,letra);
        //      printf("%ld %d %d %d \n",hilo,year,mes,dia);
        //    }
        sha256Final(&context, hash);
        for (int j = 0; j < 1715; j++) {
                sha256Init(&context);
                doubleHash(hash,context.data,32);
                context.dataLength=64;
                sha256Transform(&context, context.data);
                //doubleIntAdd(&context.bitLength[0], &context.bitLength[1], 512);
                context.bitLength[0]=512;
                context.dataLength=0;  
                sha256Final(&context, hash);
        }
       doubleHash(hash,strdata,32);
       //printf(".32s\n",strdata);
       //if (year==65)
       //if (year==96 && mes==2 && dia ==1 && letra ==8) {
         //               printf("%d\n",letra);}
                     //printHash(hash,32); }
      //               printf("%ld %d %d %d \n",hilo,year,mes,dia);}
       for (int j=0; j < 64; j++) { strdataout[j+64*hilo]=strdata[j];} 
}
}

__forceinline__ __device__ void sha256Init(sha256Context* context) {
	context->dataLength = 0;
	context->bitLength[0] = 0;
	context->bitLength[1] = 0;
	context->state[0] = 0x6a09e667;
	context->state[1] = 0xbb67ae85;
	context->state[2] = 0x3c6ef372;
	context->state[3] = 0xa54ff53a;
	context->state[4] = 0x510e527f;
	context->state[5] = 0x9b05688c;
	context->state[6] = 0x1f83d9ab;
	context->state[7] = 0x5be0cd19;
}

__forceinline__ __device__ void sha256Final(sha256Context* context, unsigned char* hash) {
	unsigned int length = context->dataLength;
        /*datalenth is zero or 19*/
	context->data[length++] = 0x80;
	for (; length < 56; length++) {
		context->data[length] = 0x00;
	}
	//	memset(context->data, 0, 56);

	// append the total message length in bits and transform.
	//doubleIntAdd(&context->bitLength[0], &context->bitLength[1],
	//		context->dataLength * 8);
#pragma unroll 2
	for (int j = 0; j < 2; j++) {
#pragma unroll 4
		for (int i = 0; i < 4; i++) {
			context->data[63 - i - 4 * j] = context->bitLength[j] >> 8 * i;
		}
	}
	sha256Transform(context, context->data);

	// implementation uses little endian byte ordering and SHA uses big endian, reverse all bytes
#pragma unroll 4
	for (int i = 0; i < 4; i++) {
#pragma unroll 8
		for (int j = 0; j < 8; j++) {
			hash[i + 4 * j] = (context->state[j] >> (24 - i * 8)) & 0x000000ff;
		}
	}
}

__device__ void sha256Transform(sha256Context* context, unsigned char* data) {
	unsigned int shadowRegister[8];
	unsigned int messageSchedule[64];

#pragma unroll 16
	for (int i = 0, j = 0; i < 16; i++, j += 4) {
		messageSchedule[i] = (data[j] << 24) | (data[j + 1] << 16)
				| (data[j + 2] << 8) | (data[j + 3]);
	}
#pragma unroll 48
	for (int i = 16; i < 64; i++) {
		messageSchedule[i] = sigma1(messageSchedule[i - 2])
				+ messageSchedule[i - 7] + sigma0(messageSchedule[i - 15])
				+ messageSchedule[i - 16];
	}

#pragma unroll 8
	for (int i = 0; i < 8; i++) {
		shadowRegister[i] = context->state[i];
	}

#pragma unroll 64
	for (int i = 0; i < 64; i++) {
		unsigned int textRegister1 = shadowRegister[7]
				+ epsilon1(shadowRegister[4])
				+ choice(shadowRegister[4], shadowRegister[5],
						shadowRegister[6]) + C[i] + messageSchedule[i];
		unsigned int textRegister2 = epsilon0(shadowRegister[0])
				+ majority(shadowRegister[0], shadowRegister[1],
						shadowRegister[2]);
#pragma unroll 7
		for (int j = 7; j > 0; j--) {
			shadowRegister[j] = shadowRegister[j - 1];
		}
		shadowRegister[0] = textRegister1 + textRegister2;
		shadowRegister[4] += textRegister1;
	}

#pragma unroll 8
	for (int i = 0; i < 8; i++) {
		context->state[i] += shadowRegister[i];
	}
}


