#include "hip/hip_runtime.h"
// C standard header files
#include <stdio.h>
#include <omp.h>
// CUDA header files
#include <hip/hip_runtime.h>

#define cudaCheck(call) __cudaCheck(call, __FILE__, __LINE__)
inline void __cudaCheck(hipError_t err, const char* file, int line) {
#ifndef NDEBUG
	if (err != hipSuccess) {
		fprintf(stderr, "%s(%d): CUDA error: %s\n", __FILE__, __LINE__,
				hipGetErrorString(err));
		exit (EXIT_FAILURE);
	}
#endif
}

#include "sha256.h"


//G969 has 2GPCs, 2*512 = 1024 cores
//1 GPC GPU Cluster has 4 SMM
//1 SMM has 4 little regions
//1 region has 4*8=32 cores
__global__ void hashtestGPU();

int myStrCmp(const void *s1, const void *s2) {
  const char *key = (char *)s1;
  //const char * const *arg = (char **) s2;
  const char *arg= (char *)s2;
   //printf("myStrCmp: s1(%p): %s, s2(%p): %.75s\n", s1, key, s2, arg);
  return strncmp(key, arg,64);
}

int main(int argc, char* argv[]) {
        //hipSetDevice();
#ifndef NDEBUG
	cudaCheck(hipDeviceReset());
#endif
        FILE *f = fopen("./somehashes.txt", "r");
        typedef char fixed_string[65];
        fixed_string *hashes;
        //char (* hashes)[65];
        hashes = (fixed_string *) malloc(5338814*65+1);
        fread(hashes,5338814*65,1,f);
        fclose(f);
        //printf("fichero leido\n");
        //printf("%.98s\n",hashes[5120929]);
        //qsort(hashes,5338814,65,(int(*)(const void *,const void*)) strcmp);
        //printf("%.98s\n",hashes[65*5120929]); 
        //unsigned char other[]="00014Z1965061308034";
    
        unsigned char *o,*h;
        unsigned char * hash;
        hash=(unsigned char *) malloc(64*31*12*90*2*23+1);

        printf("%d ",hipMalloc(&o,20));
        printf("%d \n",hipMalloc(&h,64*31*12*90*23));
    char * prov[]={"08","25", "43", "17"};
    for (int trito=0;trito<1+000;trito++){
        int dni=93359;
      //for (int dni=50000; dni <50001 ;dni++) {
        for (int pr=0; pr <2; pr++){
        char distrito[6];
        sprintf(distrito, "%.2s%03d",prov[pr],trito);
        char base[20];
        sprintf(base,"%05dP19650613%.5s",dni,distrito);
        base[19]='\0';
        // for (int l=0; l <23/23; l++) {
        char letrasDNI[] ="TRWAGMYFPDXBNJZSQVHLCKE";
        //base[5]=letrasDNI[l];
        printf("base %s\n",base);
        //unsigned char hash0[]="4efd89e2f3bb5f32e35d9249b1d90693a5a4eea69cba351e8540a1799d2d0e3b";
        cudaCheck(hipMemcpy(o, base, 20, hipMemcpyHostToDevice));
        /* K40M
           (15) Multiprocessors, (192) CUDA Cores/MP:     2880 CUDA Cores
              3*5  *32*6
              23*100/15=153.333
           <<< 15 * ....,31*12 
             192/32=6
          mejor 90 years * 23 letras.
            <<90*23,31*12>> or 45*23,31*24
        */
        //sha256cat<<<40*23,31*15*2>>>(o,h);
        dim3 threadsPerBlock(2,31*12);
        sha256cat<<<450,threadsPerBlock>>>(o,h);
        //hipMemcpy(hash,h,64*31*12*100,hipMemcpyDeviceToHost);
        //printf("output %.64s\n",&hash[64*(12+31*5+31*12*65)]);
        //printf("output %.64s\n",hash);
        //printf("compare%s\n",hash0);
        cudaCheck(hipMemcpy(hash,h,64*31*12*90*23,hipMemcpyDeviceToHost));
        #pragma omp parallel num_threads(6)
        #pragma omp for
          for (int x=0; x<31*12*90*23; x++ ) {
            //if (l==8 && x==35743) {
            char * pItem;
            pItem = (char *) bsearch(&hash[64*x],hashes,5338814,65, myStrCmp);
            if (pItem!=NULL) {
               int letra = x/(90*31*12);
               int fecha= x- letra*31*12*90;
               int year= fecha / (31*12);
               int resto=fecha - year*31*12;
               int mes = resto /31;
               int dia = resto -( mes * 31);
               mes++;
               dia++;
               year +=90;
            // if (dia==1 && mes==2 && year==96) {
               printf("%05d%c 19%02d%02d%02d %.5s encontrado %.64s %d \n", dni,letrasDNI[letra],
                       year, mes, dia, distrito, &hash[64*x],x);
             } //if
          }
        //}
      }
      }
        hipFree(o);
        hipFree(h);
        free(hashes);
	cudaCheck(hipDeviceReset());
}

