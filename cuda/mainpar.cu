#include "hip/hip_runtime.h"
// C standard header files
#include <stdio.h>
#include <omp.h>
// CUDA header files
#include <hip/hip_runtime.h>

#define cudaCheck(call) __cudaCheck(call, __FILE__, __LINE__)
inline void __cudaCheck(hipError_t err, const char* file, int line) {
#ifndef NDEBUG
	if (err != hipSuccess) {
		fprintf(stderr, "%s(%d): CUDA error: %s\n", __FILE__, __LINE__,
				hipGetErrorString(err));
		exit (EXIT_FAILURE);
	}
#endif
}

#include "sha256.h"


//G969 has 2GPCs, 2*512 = 1024 cores
//1 GPC GPU Cluster has 4 SMM
//1 SMM has 4 little regions
//1 region has 4*8=32 cores
__global__ void hashtestGPU();

int myStrCmp(const void *s1, const void *s2) {
  const char *key = (char *)s1;
  //const char * const *arg = (char **) s2;
  const char *arg= (char *)s2;
   //printf("myStrCmp: s1(%p): %s, s2(%p): %.75s\n", s1, key, s2, arg);
  return strncmp(key, arg,64);
}

int main(int argc, char* argv[]) {
        //hipSetDevice(0);
	//hashtestGPU<<<1, 1>>>();
	//cudaCheck(hipDeviceReset());
        FILE *f = fopen("./somehashes.txt", "r");
        typedef char fixed_string[65];
        fixed_string *hashes;
        //char (* hashes)[65];
        hashes = (fixed_string *) malloc(5338814*65+1);
        fread(hashes,5338814*65,1,f);
        fclose(f);
        //printf("fichero leido\n");
        //printf("%.98s\n",hashes[5120929]);
        //qsort(hashes,5338814,65,(int(*)(const void *,const void*)) strcmp);
        //printf("%.98s\n",hashes[65*5120929]); 
        //unsigned char other[]="00014Z1965061308034";
    
        unsigned char *o,*h;
        unsigned char * hash;
        hash=(unsigned char *) malloc(64*31*12*100*2*23+1);
        printf("lets allocate\n");
        hipDeviceSynchronize();
        hipDeviceSetCacheConfig(hipFuncCachePreferL1);
        printf("Cuda status: %s\n", hipGetErrorString( hipGetLastError() ) ); 
        hipMalloc(&o,20);
        printf("Cuda status: %s\n", hipGetErrorString( hipGetLastError() ) );
        printf("%d \n",hipMalloc(&h,64*31*12*90*23));
    char * prov[]={"08","25", "43", "17"};
    for (int trito=757;trito<758;trito++){
        int dni=0;
      //for (int dni=50000; dni <50001 ;dni++) {
        for (int pr=0; pr <4; pr++){
        char distrito[6];
        sprintf(distrito, "%.2s%03d",prov[pr],trito);
        char base[20];
        sprintf(base,"%05dP19650613%.5s",dni,distrito);
        base[19]='\0';
        // for (int l=0; l <23/23; l++) {
        char letrasDNI[] ="TRWAGMYFPDXBNJZSQVHLCKE";
        //base[5]=letrasDNI[l];
        printf("base %s\n",base);
        //unsigned char hash0[]="4efd89e2f3bb5f32e35d9249b1d90693a5a4eea69cba351e8540a1799d2d0e3b";
        cudaCheck(hipMemcpy(o, base, 20, hipMemcpyHostToDevice));
        //sha256cat<<<80*23,31*15>>>(o,h);
          dim3 threadsPerBlock(30,31);
          sha256cat<<<15*3,threadsPerBlock>>>(o,h); //23 ahora detro del 
        //hipMemcpy(hash,h,64*31*12*100,hipMemcpyDeviceToHost);
        //printf("output %.64s\n",&hash[64*(12+31*5+31*12*65)]);
        //printf("output %.64s\n",hash);
        //printf("compare%s\n",hash0);
        cudaCheck(hipMemcpy(hash,h,64*31*12*90*23,hipMemcpyDeviceToHost));
        #pragma omp parallel num_threads(6)
        #pragma omp for
          for (int x=0; x<31*12*90*23; x++ ) {
            //if (l==8 && x==35743) {
            char * pItem;
            pItem = (char *) bsearch(&hash[64*x],hashes,5338814,65, myStrCmp);
            if (pItem!=NULL) {
               int letra = x/(90*31*12);
               int fecha= x- letra*31*12*90;
               int year= fecha / (31*12);
               int resto=fecha - year*31*12;
               int mes = resto /31;
               int dia = resto -( mes * 31);
               mes++;
               dia++;
               year+=10;
            // if (dia==1 && mes==2 && year==96) {
               printf("%05d%c 19%02d%02d%02d %.5s encontrado %.64s %d \n", dni,letrasDNI[letra],
                       year, mes, dia, distrito, &hash[64*x],x);
             } //if
          }
        //}
      }
      }
        hipFree(o);
        hipFree(h);
        free(hashes);
	cudaCheck(hipDeviceReset());
}

